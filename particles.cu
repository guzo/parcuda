#include "hip/hip_runtime.h"
#include "particles.hpp"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>

#include <ctime>

#include <iostream>
#include <cmath>

using namespace std;

extern hipGraphicsResource_t cudavbo;

__device__ __constant__ hiprandState* curand_states;


__device__ float3 operator+(float3 a, float3 b) {
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__global__ void particles_init_gpu(float3* p, unsigned long long seed) {
	unsigned int x  = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y  = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int z  = blockIdx.z*blockDim.z + threadIdx.z;
	unsigned int i  = x + y*PX*PZ + z*PX;
	hiprandState* state = curand_states + (x + y + z)*(x%16);

	hiprand_init(seed, state - curand_states, 0, state);
	p[i*2] = make_float3(
		(float(x)/float(PX) + hiprand_normal(state)*.05f)*2.f-1.f,
		(float(y)/float(PY) + hiprand_normal(state)*.4f ),
		(float(z)/float(PZ) + hiprand_normal(state)*.05f)*2.f-1.f - 2
	);
	p[i*2+1] = make_float3(
		0,
		.002f*sin(float(z)/float(PZ)*3.14),
		.01
	);
}

__global__ void particles_step_gpu(float3* p) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;
	unsigned int i = x + y*PX*PZ + z*PX;
	hiprandState* state = curand_states + (x + y + z)*(x%16);

	float3 v = p[i*2+1] + make_float3(0.f,-.002f,0.f);
	float3 s = p[i*2  ] + v;
	if(s.y < -1) {
		v   = make_float3(v.x, -v.y, v.z);
		s.y = -1;
	}
	if(s.z > 1) {
		s.z = -1;
		s.y = .25*hiprand_normal(state);
		v.y *= 1 + .05f*hiprand_normal(state);
		if(v.y > .05)
			v.y = .002*hiprand_normal(state);
	}
	p[i*2]   = s;
	p[i*2+1] = v;
}

void particles_init(size_t particle_count) {
	size_t dummy = particle_count;
	float3* v = 0;

	float* randsd;
	WRAP_CUDA(hipMalloc(reinterpret_cast<void**>(&randsd), (PX+PY+PZ)*16*sizeof(*curand_states)));
	WRAP_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(curand_states), &randsd, sizeof(*curand_states)));

	WRAP_CUDA(hipGraphicsMapResources(1, &cudavbo));
	WRAP_CUDA(hipGraphicsResourceGetMappedPointer((void**)&v, &dummy, cudavbo));

	dim3 block(4,2,4);
	dim3 grid(PX/4,PY/2,PZ/4);
	particles_init_gpu<<<grid, block>>>(v, time(0));

	WRAP_CUDA(hipGraphicsUnmapResources(1, &cudavbo));
}

void particles_step(size_t particle_count) {
	size_t dummy = particle_count;
	float3* v = 0;

	WRAP_CUDA(hipGraphicsMapResources(1, &cudavbo));
	WRAP_CUDA(hipGraphicsResourceGetMappedPointer((void**)&v, &dummy, cudavbo));

	dim3 block(4,2,4);
	dim3 grid(PX/4,PY/2,PZ/4);
	particles_step_gpu<<<grid, block>>>(v);

	WRAP_CUDA(hipGraphicsUnmapResources(1, &cudavbo));
}
